#include "hip/hip_runtime.h"
//
// Created by arrouan on 28/09/16.
//

//#include <stdint-gcc.h>
#include <stdint.h>
#include <random>
//#include <QTZlib/zlib.h>
#include <zlib.h>
#include "Common.h"
#include <omp.h>
#include <iostream>
/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

float Common::matrix_binding_[BINDING_MATRIX_SIZE*BINDING_MATRIX_SIZE];

__global__ 
void setup_gen(hiprandState *state, uint32_t seed, int size)
{
    int y = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;
    int id = x*size+y;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ 
void proceed (float* matrix_binding, int size, int zperc,
                                hiprandState *state)
{
    float r;
    int y = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;
    int id = x*size+y;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    if (hiprand_uniform(&localState) * 100.0f > zperc) {
        r = hiprand_uniform(&localState) * 2.0f - 1.0f;
    } else {
        r = 0;
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    matrix_binding[id] = r;
}

void Common::init_binding_matrix_gpu ( uint32_t seed)
{
    hiprandState *float_gen_;
    
    int floatGenSizeInBytes = BINDING_MATRIX_SIZE *BINDING_MATRIX_SIZE* sizeof(hiprandState);
    int ok = hipMalloc((void**) &float_gen_, floatGenSizeInBytes );
    if(ok!=hipSuccess){
            std::cout << "error hipMalloc 0:" << ok << std::endl ;
            return;
    }

    float* gpuMatrixIn;
    int matrixSizeInBytes = BINDING_MATRIX_SIZE *BINDING_MATRIX_SIZE* sizeof(float);
    ok=hipMalloc((void**) &gpuMatrixIn, matrixSizeInBytes );
    if(ok!=hipSuccess){
            std::cout << "error hipMalloc 1:" << ok << std::endl ;
            return;
    }

    dim3 dimBlock(32,32);
    dim3 dimGrid(BINDING_MATRIX_SIZE/dimBlock.x, BINDING_MATRIX_SIZE/dimBlock.y);

    setup_gen<<<dimGrid,dimBlock>>> (float_gen_, seed, BINDING_MATRIX_SIZE);
    proceed<<<dimGrid,dimBlock>>> ( gpuMatrixIn,BINDING_MATRIX_SIZE,
                  BINDING_MATRIX_ZERO_PERCENT,
                  float_gen_);

    ok = hipMemcpy(matrix_binding_, gpuMatrixIn, matrixSizeInBytes,hipMemcpyDeviceToHost);
    if(ok!=hipSuccess){
            std::cout << "error hipMemcpy 2:" << ok << std::endl ;
            return;
    }

    ok = hipFree(float_gen_);
    if(ok!=hipSuccess){
            std::cout << "error hipFree 0:" << ok << std::endl ;
            return;
    }
    ok = hipFree(gpuMatrixIn);
    if(ok!=hipSuccess){
            std::cout << "error hipFree 1:" << ok << std::endl ;
            return;
    }

}

void Common::init_binding_matrix(uint32_t seed) {

  std::mt19937 float_gen_;
  float_gen_.seed(seed);
  std::uniform_real_distribution<float> dis_number(-1, 1);
  std::uniform_int_distribution<int8_t> dis_percent(0,100);

#pragma omp parallel for schedule(static)
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    #pragma omp parallel for schedule(static)
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      if (dis_percent(float_gen_) > BINDING_MATRIX_ZERO_PERCENT)
        matrix_binding_[i*BINDING_MATRIX_SIZE+j]=dis_number(float_gen_);
      else
        matrix_binding_[i*BINDING_MATRIX_SIZE+j]=0;
    }
  }

}

void Common::save_binding_matrix() {
  char* lookup_table_file_name = new char[100];

  sprintf( lookup_table_file_name, "binding_matrix.zip" );

  gzFile lookup_table_file = gzopen( lookup_table_file_name, "r" );

  if ( lookup_table_file == Z_NULL )
  {
    printf( "ERROR : Could not read lookup table file %s\n", lookup_table_file_name );
    exit( EXIT_FAILURE );
  }

  float value;
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      gzread(lookup_table_file, &value, sizeof(float));
      matrix_binding_[i*BINDING_MATRIX_SIZE+j] = value;
    }
  }

  gzclose( lookup_table_file );

  delete[] lookup_table_file_name;
}

void Common::load_binding_matrix() {
  char* lookup_table_file_name = new char[100];

  sprintf( lookup_table_file_name, "binding_matrix.zip" );

  gzFile lookup_table_file = gzopen( lookup_table_file_name, "w" );

  if ( lookup_table_file == Z_NULL )
  {
    printf( "ERROR : Could not write lookup table file %s\n", lookup_table_file_name );
    exit( EXIT_FAILURE );
  }

  double value;
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      value = matrix_binding_[i*BINDING_MATRIX_SIZE+j];

      gzwrite(lookup_table_file, &value, sizeof(float));
    }
  }

  gzclose( lookup_table_file );

  delete[] lookup_table_file_name;
}
