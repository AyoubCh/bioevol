#include "hip/hip_runtime.h"
//
// Created by arrouan on 28/09/16.
//

//#include <stdint-gcc.h>
#include <stdint.h>
#include <random>
//#include <QTZlib/zlib.h>
#include <zlib.h>
#include "Common.h"
#include <omp.h>
#include <iostream>
/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

float * Common::matrix_binding_ = (float *) calloc(BINDING_MATRIX_SIZE*BINDING_MATRIX_SIZE, sizeof(float));

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cout << "Error at " << __FILE__ << ":" << __LINE__ << std::endl; \
    return;}} while(0)

__global__ 
void setup_gen(hiprandState *state, uint32_t seed, int size)
{
    int y = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;
    int id = x*size+y;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ 
void proceed (float* matrix_binding, int size, int zperc,
                                hiprandState *state)
{
    float r;
    int y = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;
    int id = x*size+y;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    if (hiprand_uniform(&localState) * 100.0f > zperc) {
        r = hiprand_uniform(&localState) * 2.0f - 1.0f;
    } else {
        r = 0;
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    matrix_binding[id] = r;
}

void Common::init_binding_matrix_gpu ( uint32_t seed)
{
    hiprandState *float_gen_;
    
    int floatGenSizeInBytes = BINDING_MATRIX_SIZE *BINDING_MATRIX_SIZE* sizeof(hiprandState);
    CUDA_CALL(hipMalloc((void**) &float_gen_, floatGenSizeInBytes ));

    float* gpuMatrixIn;
    int matrixSizeInBytes = BINDING_MATRIX_SIZE *BINDING_MATRIX_SIZE* sizeof(float);
    CUDA_CALL(hipMalloc((void**) &gpuMatrixIn, matrixSizeInBytes ));
    dim3 dimBlock(32,32);
    dim3 dimGrid(BINDING_MATRIX_SIZE/dimBlock.x, BINDING_MATRIX_SIZE/dimBlock.y);

    setup_gen<<<dimGrid,dimBlock>>> (float_gen_, seed, BINDING_MATRIX_SIZE);
    proceed<<<dimGrid,dimBlock>>> ( gpuMatrixIn,BINDING_MATRIX_SIZE,
                  BINDING_MATRIX_ZERO_PERCENT,
                  float_gen_);

    CUDA_CALL(hipMemcpy(Common::matrix_binding_, gpuMatrixIn, matrixSizeInBytes,hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(float_gen_));
    CUDA_CALL(hipFree(gpuMatrixIn));

}

void Common::init_binding_matrix(uint32_t seed) {

  std::mt19937 float_gen_;
  float_gen_.seed(seed);
  std::uniform_real_distribution<float> dis_number(-1, 1);
  std::uniform_int_distribution<int8_t> dis_percent(0,100);

#pragma omp parallel for schedule(static)
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    #pragma omp parallel for schedule(static)
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      if (dis_percent(float_gen_) > BINDING_MATRIX_ZERO_PERCENT)
        matrix_binding_[i*BINDING_MATRIX_SIZE+j]=dis_number(float_gen_);
      else
        matrix_binding_[i*BINDING_MATRIX_SIZE+j]=0;
    }
  }

}

void Common::save_binding_matrix() {
  char* lookup_table_file_name = new char[100];

  sprintf( lookup_table_file_name, "binding_matrix.zip" );

  gzFile lookup_table_file = gzopen( lookup_table_file_name, "r" );

  if ( lookup_table_file == Z_NULL )
  {
    printf( "ERROR : Could not read lookup table file %s\n", lookup_table_file_name );
    exit( EXIT_FAILURE );
  }

  float value;
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      gzread(lookup_table_file, &value, sizeof(float));
      matrix_binding_[i*BINDING_MATRIX_SIZE+j] = value;
    }
  }

  gzclose( lookup_table_file );

  delete[] lookup_table_file_name;
}

void Common::load_binding_matrix() {
  char* lookup_table_file_name = new char[100];

  sprintf( lookup_table_file_name, "binding_matrix.zip" );

  gzFile lookup_table_file = gzopen( lookup_table_file_name, "w" );

  if ( lookup_table_file == Z_NULL )
  {
    printf( "ERROR : Could not write lookup table file %s\n", lookup_table_file_name );
    exit( EXIT_FAILURE );
  }

  double value;
  for (int i = 0; i < BINDING_MATRIX_SIZE; i++) {
    for (int j = 0; j < BINDING_MATRIX_SIZE; j++) {
      value = matrix_binding_[i*BINDING_MATRIX_SIZE+j];

      gzwrite(lookup_table_file, &value, sizeof(float));
    }
  }

  gzclose( lookup_table_file );

  delete[] lookup_table_file_name;
}
